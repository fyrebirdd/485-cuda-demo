#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "charCountGpu.hpp"

#include <iostream>

void HandleError(hipError_t cudaStatus, std::string functionCalling) {
    if (cudaStatus != hipSuccess) {
        std::cout << "CUDA ERROR: <" << functionCalling << "> " << hipGetErrorString(cudaStatus) << std::endl;
    }
}

__global__ void charCountKernel(char* chars, char characterToCount, int length, int* count){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    while (tid < length){
        if (chars[tid] == characterToCount) {
            atomicAdd(count, 1);
        }
        tid += blockDim.x * gridDim.x;
    }
}

//helper function for using CUDA to count a specific character in parallel
int charCountCuda(std::vector<char> board, char target) {
    int arraySize = board.size();
    int output = 0;

    //out
    int* dev_output = 0;
    //in
    char* dev_board = 0;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    const int blockSize = prop.maxThreadsPerBlock;
    const int gridSize = (arraySize + blockSize - 1) / blockSize;

    HandleError(hipSetDevice(0), "hipSetDevice");

    //allocate memory on gpu
    HandleError(hipMalloc((void**)&dev_output, sizeof(int)), "hipMalloc");

    HandleError(hipMalloc((void**)&dev_board, arraySize * sizeof(char)), "cudaMalloc2");
    
    //copy data to gpu
    HandleError(hipMemcpy(dev_board, board.data(), arraySize * sizeof(char), hipMemcpyHostToDevice), "hipMemcpy Pre");

    //launch kernel
    charCountKernel <<<gridSize, blockSize>>> (dev_board, target, arraySize, dev_output);

    // blocks main thread until kernel is finished, returns any errors
    HandleError(hipDeviceSynchronize(), "hipDeviceSynchronize");

    HandleError(hipGetLastError(), "hipGetLastError");

    HandleError(hipMemcpy(&output, dev_output, sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy Post");

    hipFree(dev_output);
    hipFree(dev_board);

    return output;
}
