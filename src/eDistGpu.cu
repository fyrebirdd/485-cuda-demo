#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "eDistGpu.hpp"

#include <iostream>

void HandleError(hipError_t cudaStatus, std::string functionCalling) {
    if (cudaStatus != hipSuccess) {
        std::cout << "CUDA ERROR: <" << functionCalling << "> " << hipGetErrorString(cudaStatus) << std::endl;
    }
}

__global__ void euclideanDistanceKernel(char* vector, int size, float* result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        int valueAsInt = (vector[tid] - '0');
        float square = (float)(valueAsInt * valueAsInt);
        atomicAdd(result,square);
    }
}

float euclideanDistanceCUDA(std::vector<char>& inputVec) {
    int size = static_cast<int>(inputVec.size());

    // Device vectors
    char* d_vector;
    float* d_result;
    float h_result = 0;

    HandleError(hipSetDevice(0), "hipSetDevice");

    // Allocate memory on the device
    HandleError(hipMalloc((void**)&d_vector, size * sizeof(char)), "hipMalloc");
    HandleError(hipMalloc((void**)&d_result, sizeof(float)), "cudaMalloc2");

    // Copy input vector from host to device
    HandleError(hipMemcpy(d_vector, inputVec.data(), size * sizeof(char), hipMemcpyHostToDevice), "hipMemcpy pre");

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    // Define grid and block sizes
    int blockSize = prop.maxThreadsPerBlock;
    int gridSize = (size + blockSize - 1) / blockSize;

    // Launch the kernel to convert characters to floats
    euclideanDistanceKernel<<<gridSize, blockSize>>>(d_vector, size, d_result);

    HandleError(hipDeviceSynchronize(), "hipDeviceSynchronize");

    HandleError(hipGetLastError(), "hipGetLastError");

    // Copy the result vector from device to host
    HandleError(hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy post");
    // Free memory on the device
    hipFree(d_vector);
    hipFree(d_result);

    return sqrt(h_result);
}